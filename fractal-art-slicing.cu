#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#define H_RES 3840 // horizontal resolution
#define V_RES 2160 // vertical resolution
#define CENTER_X 0 // X coordinate for image center
#define CENTER_Y 0 // Y coordinate for image center
#define SCALE 2 // maximum X value in the fractal graph
#define ITERATIONS (1 << 8) // number of iteration for checking divergence
#define R 2 // ceiling upon which function is considered divergent
#define SHADOW_DISTANCE 64 // radius of the circular shadow plot
#define SHADOW_SHARPNESS 1 // rapidity with which shadow gets dark
#define SHADOW_TILT_H -64 // horizontal offset from where shadow is plotted
#define SHADOW_TILT_V 32 // vertical offset from where shadow is plotted
#define SHADOW_INTENSITY 0.8 // blackness of the shadow
#define BLOCK_DIM 16 // threads per block dimension
#define OUT 0xFF // outside color of the fractal mask
#define IN 0x00 // inside color of the fractal mask
#define H_EXTENSION (abs(SHADOW_TILT_H) + SHADOW_DISTANCE) // horizontal extension due to shadow offset
#define V_EXTENSION (abs(SHADOW_TILT_V) + SHADOW_DISTANCE) // vertical extension due to shadow offset
#define H_EXTENDED (H_RES + 2 * H_EXTENSION) // required vertical dimension due to shadow offset
#define V_EXTENDED (V_RES + 2 * V_EXTENSION) // required horizontal dimension due to shadow offset
#define MASK_COORDINATES(x, y) ((y) * H_EXTENDED + (x)) // linearized coordinates of mask
#define SHADOW_COORDINATES(x, y) ((y) * H_EXTENDED + (x)) // linearized coordinates of shadow
#define IMAGE_COORDINATES(x, y) ((y) * H_RES + (x)) // linearized coordinates of images
#define SLICES (2 * (SHADOW_DISTANCE + BLOCK_DIM - 1) / BLOCK_DIM + 1) // number of slices needed to avoid memory collisions

typedef unsigned char byte;
typedef struct complex {
    double r;
    double i;
} complex;

/// Check the last kernel call for errors
#define CHECK_KERNELCALL {                                  \
    const hipError_t err = hipGetLastError();             \
    if (err != hipSuccess) {                               \
        printf("ERROR: %s::%d\n -> %s\n",                   \
            __FILE__, __LINE__, hipGetErrorString(err));   \
        exit(EXIT_FAILURE);                                 \
    }                                                       \
}

/// Calculate fractal and shadow for each pixel point and assign images respectively
__host__ void generate_art(const complex *c, byte *image, const byte *inside, const byte *outside);

/// The first iteration generates a black and white image (mask) where
/// pixels inside (divergent) the fractal are black and pixels outside
/// (convergent) are white.
__global__ void compute_mask(
    const complex c,
    byte *__restrict__ mask);

/// The second iteration plots a circular shadow for each outside pixel of
/// the just generated fractal mask.
/// This is done by adding one to the corresponding elements of the shadow
/// integer array (sized like fractal mask).
/// The higher is the number, the higher is the shadow intensity.
__global__ void apply_shadow(
    const int h_slice,
    const int v_slice,
    const byte *__restrict__ mask,
    int *__restrict__ shadow);

/// The third iteration assigns the inside and the outside images.
/// The shadow toner for the inner image is computed starting from the corresponding
/// value in the shadow array.
__global__ void assign_final(
    const int *__restrict__ shadow,
    const byte *__restrict__ mask,
    const byte *__restrict__ inside,
    const byte *__restrict__ outside,
    byte *__restrict__ image);

/// Complex multiplication
__device__ void cmul(complex *outcome, const complex *first, const complex *second);

/// Complex sum
__device__ void csum(complex *outcome, const complex *first, const complex *second);

/// Complex absolute value
__device__ double cmod(const complex *z);

/// Save ppm image on disk
int save_image(const char *filename, unsigned char *image);

/// Load ppm image from disk
int load_image(const char *filename, unsigned char *image);

/// Measure milliseconds on host
double milliseconds();

int main(int argc, char **argv) {

    // Retrieve c constant from input args
    if (argc != 3) {
        printf("Provide a complex number");
        return 1;
    }
    complex c;
    c.r = strtod(argv[1], NULL);
    c.i = strtod(argv[2], NULL);

    // Allocate memory for images
    byte *image = (byte *)malloc(3 * V_RES * H_RES * sizeof(byte));
    byte *inside = (byte *)malloc(3 * V_RES * H_RES * sizeof(byte));
    byte *outside = (byte *)malloc(3 * V_RES * H_RES * sizeof(byte));

    // Load the two input images
    if (load_image("inside.ppm", inside) < 0) {
        fprintf(stderr, "Error opening %s\n", "inside.ppm");
        return 1;
    }
    if (load_image("outside.ppm", outside) < 0) {
        fprintf(stderr, "Error opening %s\n", "outside.ppm");
        return 1;
    }

    // Compute fractal, shadow and image assignment
    generate_art(&c, image, inside, outside);

    // Save the output image
    if (save_image("fractal.ppm", image) < 0) {
        fprintf(stderr, "Error opening %s\n", "fractal.ppm");
        return 1;
    }

    // Free images memory
    free(image);
    free(inside);
    free(outside);
    return 0;
}

__host__ void generate_art(const complex *c, byte *image, const byte *inside, const byte *outside) {

    // Initialize events
    float time;
    double start_ms, stop_ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory
    byte *mask_d, *inside_d, *outside_d, *image_d;
    int *shadow_d;
    hipMalloc(&mask_d, V_EXTENDED * H_EXTENDED * sizeof(byte));
    hipMalloc(&shadow_d, V_EXTENDED * H_EXTENDED * sizeof(int));
    hipMalloc(&inside_d, 3 * V_RES * H_RES * sizeof(byte));
    hipMalloc(&outside_d, 3 * V_RES * H_RES * sizeof(byte));
    hipMalloc(&image_d, 3 * V_RES * H_RES * sizeof(byte));

    // Data initialization
    hipMemset(mask_d, 0x00, V_EXTENDED * H_EXTENDED * sizeof(byte));
    hipMemset(shadow_d, 0, V_EXTENDED * H_EXTENDED * sizeof(int));

    // Data transfer to device
    start_ms = milliseconds();
    hipMemcpy(inside_d, inside, 3 * V_RES * H_RES * sizeof(byte), hipMemcpyHostToDevice);
    hipMemcpy(outside_d, outside, 3 * V_RES * H_RES * sizeof(byte), hipMemcpyHostToDevice);
    stop_ms = milliseconds();
    printf("Memory transfer to device: %f\n", stop_ms - start_ms);

    dim3 block_size, grid_size;
    block_size = dim3(BLOCK_DIM, BLOCK_DIM);

    // For each pixel compute fractal mask
    hipEventRecord(start);
    grid_size = dim3(
        ceil((float)H_EXTENDED / block_size.x),
        ceil((float)V_EXTENDED / block_size.y));
    compute_mask << <grid_size, block_size >> > (*c, mask_d);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    CHECK_KERNELCALL;
    hipEventElapsedTime(&time, start, stop);
    printf("Mask computation: %f\n", time);

    // For each pixel compute shadow value
    hipEventRecord(start);
    for (int i = 0; i < SLICES; i++)
        for (int j = 0; j < SLICES; j++) {
            grid_size = dim3(
                ceil((float)(round((float)H_EXTENDED / block_size.x) - i) / SLICES),
                ceil((float)(round((float)V_EXTENDED / block_size.y) - j) / SLICES));
            apply_shadow << <grid_size, block_size >> > (i, j, mask_d, shadow_d);
            hipDeviceSynchronize();
            CHECK_KERNELCALL;
        }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    printf("Shadow application: %f\n", time);

    // For each pixel select final image, computing its shadow
    hipEventRecord(start);
    grid_size = dim3(
        ceil((float)H_RES / block_size.x),
        ceil((float)V_RES / block_size.y));
    assign_final << <grid_size, block_size >> > (shadow_d, mask_d, inside_d, outside_d, image_d);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    CHECK_KERNELCALL;
    hipEventElapsedTime(&time, start, stop);
    printf("Final assignment: %f\n", time);

    // Data transfer to host
    start_ms = milliseconds();
    hipMemcpy(image, image_d, 3 * V_RES * H_RES * sizeof(byte), hipMemcpyDeviceToHost);
    stop_ms = milliseconds();
    printf("Memory transfer to host: %f\n", stop_ms - start_ms);

    // Free the allocated memory
    hipFree(mask_d);
    hipFree(shadow_d);
    hipFree(inside_d);
    hipFree(outside_d);
    hipFree(image_d);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

__global__ void compute_mask(
    const complex c,
    byte *__restrict__ mask) {

    // Calculate coordinates of the pixel
    int h = blockIdx.x * blockDim.x + threadIdx.x;
    int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (h >= H_EXTENDED || v >= V_EXTENDED) return;

    // Calculate the side length of a pixel in the complex plane
    double res_unit = (double)SCALE / (H_RES / 2);

    // Calculate coordinates of the pixel in the complex plane
    complex z0, z1;
    z0.r = res_unit * (h - H_EXTENDED / 2) + CENTER_X;
    z0.i = res_unit * (v - V_EXTENDED / 2) + CENTER_Y;

    // Iterate the function on itself to then analyze the convergence
    for (int i = 0; i < ITERATIONS; i++) {

        // Compute function z1 = z0^2 + c
        cmul(&z1, &z0, &z0);
        csum(&z1, &z1, &c);
        z0.r = z1.r;
        z0.i = z1.i;

        // Check if function has diverged
        if (cmod(&z0) > R) {

            // Assign outside value
            mask[MASK_COORDINATES(h, v)] = OUT;
            return;
        }
    }
}

__global__ void apply_shadow(
    const int h_slice,
    const int v_slice,
    const byte *__restrict__ mask,
    int *__restrict__ shadow) {
#define SHADOW_TILE_DIM (BLOCK_DIM + 2 * SHADOW_DISTANCE) // shared shadow matrix side length

    // Calculate coordinates of the pixel
    int h = (SLICES * blockIdx.x + h_slice) * blockDim.x + threadIdx.x;
    int v = (SLICES * blockIdx.y + v_slice) * blockDim.y + threadIdx.y;

    // Allocate and intialize shared space
    __shared__ unsigned short shadow_tile[SHADOW_TILE_DIM][SHADOW_TILE_DIM];
    for (int i = threadIdx.x; i < SHADOW_TILE_DIM; i += BLOCK_DIM)
        for (int j = threadIdx.y; j < SHADOW_TILE_DIM; j += BLOCK_DIM)
            shadow_tile[i][j] = 0;

    // Check boundaries and ignore points in the image below
    bool plot = h < H_EXTENDED && v < V_EXTENDED && mask[MASK_COORDINATES(h, v)] == OUT;

    // Plot a circular shadow
    for (int i = -SHADOW_DISTANCE; i <= SHADOW_DISTANCE; i++) {
        for (int j = -SHADOW_DISTANCE; j <= SHADOW_DISTANCE; j++) {
            __syncthreads();

            // Increment shadow value if the current shadow index is inside borders and radius
            if (plot && h + i < H_EXTENDED && h + i >= 0 && v + j < V_EXTENDED && v + j >= 0 &&
                i * i + j * j < SHADOW_DISTANCE * SHADOW_DISTANCE)
                shadow_tile[SHADOW_DISTANCE + threadIdx.x + i][SHADOW_DISTANCE + threadIdx.y + j]++;
        }
    }

    __syncthreads();

    // Update global memory with shadow values
    for (int i = threadIdx.x; i < SHADOW_TILE_DIM; i += BLOCK_DIM)
        for (int j = threadIdx.y; j < SHADOW_TILE_DIM; j += BLOCK_DIM) {
            int x = h - threadIdx.x - SHADOW_DISTANCE + i;
            int y = v - threadIdx.y - SHADOW_DISTANCE + j;
            if (x >= 0 && x < H_EXTENDED && y >= 0 && y < V_EXTENDED)
                shadow[SHADOW_COORDINATES(x, y)] += shadow_tile[i][j];
        }

#undef SHADOW_TILE_DIM
}

__global__ void assign_final(
    const int *__restrict__ shadow,
    const byte *__restrict__ mask,
    const byte *__restrict__ inside,
    const byte *__restrict__ outside,
    byte *__restrict__ image) {

    // Calculate coordinates of the pixel
    int h = blockIdx.x * blockDim.x + threadIdx.x;
    int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (h >= H_RES || v >= V_RES) return;

    // Calculate index of the pixel
    int image_idx = IMAGE_COORDINATES(h, v);

    if (mask[MASK_COORDINATES(H_EXTENSION + h, V_EXTENSION + v)] == OUT) {

        // Outside image assignment
        image[3 * image_idx + 0] = outside[3 * image_idx + 0];
        image[3 * image_idx + 1] = outside[3 * image_idx + 1];
        image[3 * image_idx + 2] = outside[3 * image_idx + 2];

    } else {

        // Shadow intensity computation
        float toner = (exp(-SHADOW_SHARPNESS *
            shadow[SHADOW_COORDINATES(
                H_EXTENSION + h + (SHADOW_TILT_H),
                V_EXTENSION + v + (SHADOW_TILT_V))] /
            (3.1416 * SHADOW_DISTANCE * SHADOW_DISTANCE))) *
            SHADOW_INTENSITY + (1 - SHADOW_INTENSITY);

        // Inside image assignment with shadow
        image[3 * image_idx + 0] = inside[3 * image_idx + 0] * toner;
        image[3 * image_idx + 1] = inside[3 * image_idx + 1] * toner;
        image[3 * image_idx + 2] = inside[3 * image_idx + 2] * toner;

    }
}

__device__ inline void cmul(complex *outcome, const complex *first, const complex *second) {
    outcome->r = first->r * second->r - first->i * second->i;
    outcome->i = first->r * second->i + first->i * second->r;
}

__device__ inline void csum(complex *outcome, const complex *first, const complex *second) {
    outcome->r = first->r + second->r;
    outcome->i = first->i + second->i;
}

__device__ inline double cmod(const complex *z) {
    return sqrt(z->r * z->r + z->i * z->i);
}

int save_image(const char *filename, unsigned char *image) {
    FILE *f = fopen(filename, "wb");
    if (f == NULL) return -1;
    fprintf(f, "P6\n%d %d\n%d\n", H_RES, V_RES, 255);
    fwrite(image, sizeof(unsigned char), H_RES * V_RES * 3, f);
    fclose(f);
    return 0;
}

int load_image(const char *filename, unsigned char *image) {
    FILE *f = fopen(filename, "rb");
    if (f == NULL) return -1;
    char temp1[4];
    int temp2, h, v;
    fscanf(f, "%s\n%d %d\n%d\n", temp1, &h, &v, &temp2);
    if (h != H_RES || v != V_RES) {
        fclose(f);
        return -1;
    }
    fread(image, sizeof(unsigned char), H_RES * V_RES * 3, f);
    fclose(f);
    return 0;
}

inline double milliseconds() {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1000 + t.tv_usec * 0.001;
}
