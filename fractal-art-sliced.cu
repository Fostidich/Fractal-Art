#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#define H_RES 3840 // horizontal resolution
#define V_RES 2160 // vertical resolution
#define CENTER_X 0 // X coordinate for image center
#define CENTER_Y 0 // Y coordinate for image center
#define SCALE 2 // maximum X value in the fractal graph
#define ITERATIONS (1 << 8) // number of iteration for checking divergence
#define R (1 << 8) // ceiling upon which function is considered divergent
#define SHADOW_DISTANCE 16 // radius of the circular shadow plot
#define SHADOW_SHARPNESS 1 // rapidity with which shadow gets dark
#define SHADOW_TILT_H -64 // horizontal offset from where shadow is plotted
#define SHADOW_TILT_V 32 // vertical offset from where shadow is plotted
#define SHADOW_INTENSITY 0.8 // blackness of the shadow

typedef unsigned char byte;
typedef struct complex {
    double r;
    double i;
} complex;

/// Calculate fractal and shadow for each pixel point and assign images respectively
__host__ void generate_fractal(const complex *c, byte *image, const byte *inside, const byte *outside);

/// Complex multiplication
__device__ void cmul(complex *outcome, const complex *first, const complex *second);

/// Complex sum
__device__ void csum(complex *outcome, const complex *first, const complex *second);

/// Complex absolute value
__device__ double cmod(const complex *z);

/// Save ppm image on disk
int save_image(const char *filename, unsigned char *image);

/// Load ppm image from disk
int load_image(const char *filename, unsigned char *image);

/// Measure milliseconds
double milliseconds();

int main(int argc, char **argv) {

    // Retrieve c constant from input args
    if (argc != 3) {
        printf("Provide a complex number");
        return 1;
    }
    complex c;
    c.r = strtod(argv[1], NULL);
    c.i = strtod(argv[2], NULL);

    // Allocate memory for input and output images
    byte *image = (byte *)malloc(3 * V_RES * H_RES * sizeof(byte));
    byte *inside = (byte *)malloc(3 * V_RES * H_RES * sizeof(byte));
    byte *outside = (byte *)malloc(3 * V_RES * H_RES * sizeof(byte));

    // Load the two input images
    if (load_image("inside.ppm", inside) < 0) {
        fprintf(stderr, "Error opening %s\n", "inside.ppm");
        return 1;
    }
    if (load_image("outside.ppm", outside) < 0) {
        fprintf(stderr, "Error opening %s\n", "outside.ppm");
        return 1;
    }

    // Compute fractal, shadow and image assignment
    generate_fractal(&c, image, inside, outside);

    // Save the output image
    if (save_image("fractal.ppm", image) < 0) {
        fprintf(stderr, "Error opening %s\n", "fractal.ppm");
        return 1;
    }

    // Free images memory
    free(image);
    free(inside);
    free(outside);
    return 0;
}

#define OUT 0xFF // outside color of the fractal mask
#define IN 0x00 // inside color of the fractal mask

/// The first iteration generates a black and white image (mask) where
/// pixels inside the fractal are black and pixels outside are white.
__global__ void __compute_mask(
    int h_max,
    int v_max,
    const complex c,
    byte *__restrict__ mask);

/// The second iteration plots a circular shadow for each white pixel of
/// the just generated fractal mask.
/// This is done by adding one to the corresponding elements of the shadow
/// integer array (sized like fractal mask).
/// The higher is the number, the higher is the shadow intensity.
__global__ void __apply_shadow(
    int h_max,
    int v_max,
    const byte *__restrict__ mask,
    int *__restrict__ shadow);

/// The third iteration assigns the inside and the outside images.
/// The shadow toner for the inner image is computed starting from the corresponding
/// value in the shadow array.
__global__ void __assign_final(
    const int *__restrict__ shadow,
    const byte *__restrict__ mask,
    const byte *__restrict__ inside,
    const byte *__restrict__ outside,
    byte *__restrict__ image);

__host__ void generate_fractal(const complex *c, byte *image, const byte *inside, const byte *outside) {
    // Block dimensions for kernels
#define BLOCK_DIM 16
    // Required vertical dimension due to shadow offset
#define H_EXTENDED (H_RES + 2 * (abs(SHADOW_TILT_H) + SHADOW_DISTANCE))
    // Required horizontal dimension due to shadow offset
#define V_EXTENDED (V_RES + 2 * (abs(SHADOW_TILT_V) + SHADOW_DISTANCE))

    // Initialize events
    float time;
    double start_ms, stop_ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory
    byte *mask_d, *inside_d, *outside_d, *image_d;
    int *shadow_d;
    hipMalloc(&mask_d, V_EXTENDED * H_EXTENDED * sizeof(byte));
    hipMalloc(&shadow_d, V_EXTENDED * H_EXTENDED * sizeof(int));
    hipMalloc(&inside_d, 3 * V_RES * H_RES * sizeof(byte));
    hipMalloc(&outside_d, 3 * V_RES * H_RES * sizeof(byte));
    hipMalloc(&image_d, 3 * V_RES * H_RES * sizeof(byte));

    // Data initialization
    hipMemset(mask_d, 0x00, V_EXTENDED * H_EXTENDED * sizeof(byte));
    hipMemset(shadow_d, 0, V_EXTENDED * H_EXTENDED * sizeof(int));

    // Data transfer to device
    start_ms = milliseconds();
    hipMemcpy(inside_d, inside, 3 * V_RES * H_RES * sizeof(byte), hipMemcpyHostToDevice);
    hipMemcpy(outside_d, outside, 3 * V_RES * H_RES * sizeof(byte), hipMemcpyHostToDevice);
    stop_ms = milliseconds();
    printf("Memory transfer to device: %f\n", stop_ms - start_ms);

    dim3 block_size, grid_size;
    block_size = dim3(BLOCK_DIM, BLOCK_DIM);

    // For each pixel compute fractal mask
    hipEventRecord(start);
    grid_size = dim3(ceil(H_EXTENDED / block_size.x), ceil(V_EXTENDED / block_size.y));
    __compute_mask << <grid_size, block_size >> > (H_EXTENDED, V_EXTENDED, *c, mask_d);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    printf("Mask computation: %f\n", time);

    // For each pixel compute shadow value
    hipEventRecord(start);
    for (int i = 0; i < 4; i++) {
        grid_size = dim3(
            (ceil(H_EXTENDED / block_size.x) + i % 2) / 2,
            (ceil(V_EXTENDED / block_size.y) + i / 2 % 2) / 2);
        __apply_shadow << <grid_size, block_size >> > (i, mask_d, shadow_d);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    printf("Shadow application: %f\n", time);

    // For each pixel select final image, computing its shadow
    hipEventRecord(start);
    grid_size = dim3(ceil(H_RES / block_size.x), ceil(V_RES / block_size.y));
    __assign_final << <grid_size, block_size >> > (shadow_d, mask_d, inside_d, outside_d, image_d);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    printf("Final assignment: %f\n", time);

    // Data transfer to host
    start_ms = milliseconds();
    hipMemcpy(image, image_d, 3 * V_RES * H_RES * sizeof(byte), hipMemcpyDeviceToHost);
    stop_ms = milliseconds();
    printf("Memory transfer to host: %f\n", stop_ms - start_ms);

    // Free the allocated memory
    hipFree(mask_d);
    hipFree(shadow_d);
    hipFree(inside_d);
    hipFree(outside_d);
    hipFree(image_d);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

#undef H_EXTENDED
#undef V_EXTENDED
#undef BLOCK_DIM
}

__global__ void __compute_mask(
    int h_max,
    int v_max,
    const complex c,
    byte *__restrict__ mask) {

    // Calculate index of the pixel
    int h = blockIdx.x * blockDim.x + threadIdx.x;
    int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (h >= h_max || v >= v_max) return;
    int idx = v * h_max + h;

    // Calculate the side length of a pixel in the complex plane
    double res_unit = (double)SCALE / (H_RES / 2);

    // Calculate coordinates of the pixel in the complex plane
    complex z0, z1;
    z0.r = res_unit * (h - h_max / 2) + CENTER_X;
    z0.i = res_unit * (v - v_max / 2) + CENTER_Y;

    // Iterate the function on itself to then analyze the convergence
    for (int i = 0; i < ITERATIONS; i++) {

        // Compute function z1 = z0^2 + c
        cmul(&z1, &z0, &z0);
        csum(&z1, &z1, &c);
        z0.r = z1.r;
        z0.i = z1.i;

        // Check if function has diverged
        if (cmod(&z0) > R) {

            // Assign outside value
            mask[idx] = OUT;
            return;
        }
    }
}

__global__ void __apply_shadow(
    int h_max,
    int v_max,
    __restrict__ const byte *mask,
    int *__restrict__ shadow) {

    // Calculate index of the pixel
    int h = blockIdx.x * blockDim.x + threadIdx.x;
    int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (h >= h_max || v >= v_max) return;
    int idx = v * h_max + h;

    // Ignore points in the image below
    if (mask[idx] == IN) return;

    // Plot a shadow circle
#pragma unroll
    for (int i = -SHADOW_DISTANCE; i < SHADOW_DISTANCE; i++) {
#pragma unroll
        for (int j = -SHADOW_DISTANCE; j < SHADOW_DISTANCE; j++) {
            __syncthreads();

            // Calculate index of the offset shadow
            int shadow_idx = (v + j + SHADOW_TILT_V) * h_max + h + i + SHADOW_TILT_H;

            // Check that the current shadow index is inside borders and radius
            if (shadow_idx < 0 || shadow_idx >= h_max * v_max ||
                sqrt(pow(idx % h_max - shadow_idx % h_max + SHADOW_TILT_H, 2) +
                    pow(idx / h_max - shadow_idx / h_max + SHADOW_TILT_V, 2)) > SHADOW_DISTANCE)
                continue;

            atomicAdd(&shadow[shadow_idx], 1);
        }
    }
}

__global__ void __assign_final(
    const int *__restrict__ shadow,
    const byte *__restrict__ mask,
    const byte *__restrict__ inside,
    const byte *__restrict__ outside,
    byte *__restrict__ image) {

    // Calculate index of the pixel
    int h = blockIdx.x * blockDim.x + threadIdx.x;
    int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (h >= H_RES || v >= V_RES) return;
    int idx = v * H_RES + h;
    int idx_framed = (abs(SHADOW_TILT_V) + SHADOW_DISTANCE + v) *
        (H_RES + 2 * (abs(SHADOW_TILT_H) + SHADOW_DISTANCE)) +
        abs(SHADOW_TILT_H) + SHADOW_DISTANCE + h;

    if (mask[idx_framed] == OUT) {

        // Outside image assignment
        image[3 * idx + 0] = outside[3 * idx + 0];
        image[3 * idx + 1] = outside[3 * idx + 1];
        image[3 * idx + 2] = outside[3 * idx + 2];

    } else {

        // Shadow intensity computation
        float toner = (exp(-SHADOW_SHARPNESS * shadow[idx_framed] /
            (3.1416 * SHADOW_DISTANCE * SHADOW_DISTANCE))) *
            SHADOW_INTENSITY + (1 - SHADOW_INTENSITY);

        // Inside image assignment with shadow
        image[3 * idx + 0] = inside[3 * idx + 0] * toner;
        image[3 * idx + 1] = inside[3 * idx + 1] * toner;
        image[3 * idx + 2] = inside[3 * idx + 2] * toner;
    }
}

#undef IN
#undef OUT

__device__ inline void cmul(complex *outcome, const complex *first, const complex *second) {
    outcome->r = first->r * second->r - first->i * second->i;
    outcome->i = first->r * second->i + first->i * second->r;
}

__device__ inline void csum(complex *outcome, const complex *first, const complex *second) {
    outcome->r = first->r + second->r;
    outcome->i = first->i + second->i;
}

__device__ inline double cmod(const complex *z) {
    return sqrt(z->r * z->r + z->i * z->i);
}

int save_image(const char *filename, unsigned char *image) {
    FILE *f = fopen(filename, "wb");
    if (f == NULL) return -1;
    fprintf(f, "P6\n%d %d\n%d\n", H_RES, V_RES, 255);
    fwrite(image, sizeof(unsigned char), H_RES * V_RES * 3, f);
    fclose(f);
    return 0;
}

int load_image(const char *filename, unsigned char *image) {
    FILE *f = fopen(filename, "rb");
    if (f == NULL) return -1;
    char temp1[4];
    int temp2, h, v;
    fscanf(f, "%s\n%d %d\n%d\n", temp1, &h, &v, &temp2);
    if (h != H_RES || v != V_RES) {
        fclose(f);
        return -1;
    }
    fread(image, sizeof(unsigned char), H_RES * V_RES * 3, f);
    fclose(f);
    return 0;
}

inline double milliseconds() {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1000 + t.tv_usec * 0.001;
}
